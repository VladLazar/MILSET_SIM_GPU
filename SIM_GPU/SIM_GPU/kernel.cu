#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/count.h>
#include <thrust/transform.h>
#include <thrust/random.h>
#include <iostream>
#include <time.h>
#include <utility>
#include <cstdio>
#include <iomanip>
#include <fstream>
#include <thrust/tuple.h>

int N = 1000;


struct Point{
	long double x, y, z;
};

struct randomPoint{ //generez puncte random intre 0 si 1.0
	__host__ __device__ thrust::tuple<double, double, double, double, double, double> operator()(const unsigned int n){
		thrust::default_random_engine rng; // fac generatoru
		rng.discard(n); //???
		return thrust::make_tuple(
			(double)rng() / thrust::default_random_engine::max,//x
			(double)rng() / thrust::default_random_engine::max,//y
			(double)rng() / thrust::default_random_engine::max,//aux pt unghi
			(double)rng() / thrust::default_random_engine::max,//alt aux
			(double)rng() / (thrust::default_random_engine::max / 0.87) + 0.174,//unghi mic de bounce
			(double)rng() / (thrust::default_random_engine::max / 2.094) + 1.047);//unghi mare de bounce
	}
};


struct functor{

	const float3 nucleus = make_float3(0.5, 0.5, 100);
	const int distAtomCerc = 50;
	const double razaAtomuluiAur = 0.7;
	const long long distAtomPlan = 100000;
	const int Zp = 2;
	const int ZAu = 79;//for the nucleus
	const long double e = 1.602;
	const int v = 15; // m/s v standard = 15000000 m/s; val * 10 ^ 6
	const long double epsilon0 = 8.85;
	const long double pi = 3.14;//add more
	const long double alphaMass = 6.644; //in kg
	const long double eps = 0.0000000000001;

	__host__ __device__ thrust::tuple<double, double, double, double> operator()(thrust::tuple<double, double, double, double, double, double> punct){
		long double x = thrust::get<0>(punct);
		long double y = thrust::get<1>(punct);
		long double z = 0;
		long double w = thrust::get<2>(punct);

		long double b = sqrt((x - nucleus.x) * (x - nucleus.x) + (y - nucleus.y) * (y - nucleus.y));
		//x >= 0.45 && x <= 0.55 && y >= 0.45 && y <= 0.55
		if (b < 0.1)
		{
			if (w < 0.7)
			{
				long double kk = thrust::get<4>(punct);

				Point A, B;
				double radiusOfCircle = distAtomCerc * tan(kk);
				long double angle = thrust::get<3>(punct) * 2 * pi;

				A.z = nucleus.z;
				A.x = x;
				A.y = y;
				B.z = nucleus.z + distAtomCerc;
				//B.x = x + cos(angle * pi / 180.0);
				//B.y = y + sin(angle * pi / 180.0);
				B.x = x + cos(angle) * radiusOfCircle;
				B.x = y + sin(angle) * radiusOfCircle;

				long double t = ((nucleus.z + distAtomPlan) - A.z) / (B.z - A.z); //parameter;

				return thrust::make_tuple(
					(double)A.x + (B.x - A.x) * t,
					(double)A.y + (B.y - A.y) * t,
					(double)nucleus.z + distAtomPlan,
					(double)(kk * 180) / pi);
			}
			else
			{
				long double kk = thrust::get<5>(punct);
				return thrust::make_tuple(0, 0, 0, (kk * 180) / pi);
			}
		}

		//long double tg2 = (Zp * ZAu * e * e / (b * 2 * pi * epsilon0 * alphaMass * v * v)) * pow(10.0, 4);//tg (unghi de dev / 2)
		double tg2 = 0.0004993 / b;
		double tg = 2 * tg2 / (1 - tg2 * tg2);//tg deviation angle
		double angle = w * 2 * pi;

		//if (w > 0.5) tg = fabs(tg);

		double radiusOfCircle = distAtomCerc * tg;

		if (distAtomCerc <= distAtomPlan)
		{
			Point A, B;

			A.z = nucleus.z;
			A.x = x;
			A.y = y;
			B.z = nucleus.z + distAtomCerc;
			B.x = x + cos(angle) * radiusOfCircle;
			B.y = y + sin(angle) * radiusOfCircle;

			double t = ((nucleus.z + distAtomPlan) - A.z) / (B.z - A.z); //parameter;

			return thrust::make_tuple(
				(double)A.x + (B.x - A.x) * t,
				(double)A.y + (B.y - A.y) * t,
				(double)nucleus.z + distAtomPlan,
				(double)atan(tg) * 180 / pi);
		}
		else
		{
			return thrust::make_tuple(69, 69, 69, 69);//the reflected atom does not hit the screen
		}
	}
};

int main()
{
	//std::cin >> N;

	thrust::device_vector< thrust::tuple<double, double, double, double, double, double> > particles(N);
	thrust::device_vector< thrust::tuple<double, double, double, double> > intersectPlan(N);
	thrust::counting_iterator<unsigned int> counter(0);
	thrust::transform(counter, counter + N, particles.begin(), randomPoint());

	thrust::transform(particles.begin(), particles.end(), intersectPlan.begin(), functor());

	//std::cout << std::setprecision(3) << std::fixed;

	for (int i = 0; i < intersectPlan.size(); ++i)
	{
		thrust::tuple<double, double, double, double> aux = intersectPlan[i];
		std::cout << "NUM: " << i + 1 << " " << thrust::get<0>(aux) << " " << thrust::get<1>(aux) << " " << thrust::get<2>(aux) << " " << thrust::get<3>(aux) << "\n";
	}

	std::cout << "\n\n-----> DATA FOR GRAPH <-----\n\n";

	for (int i = 0; i < intersectPlan.size(); ++i)
	{
		thrust::tuple<double, double, double, double> aux = intersectPlan[i];
		std::cout << std::setiosflags(std::ios::fixed) << thrust::get<0>(aux) << ',' << thrust::get<1>(aux) << "\n";
	}

	std::cout << "\n\n-----> DATA FOR GRAPH2 <-----\n\n";

	for (int i = 0; i < particles.size(); ++i)
	{
		thrust::tuple<double, double, double, double, double, double> aux = particles[i];
		std::cout << std::setiosflags(std::ios::fixed) << thrust::get<0>(aux) << ',' << thrust::get<1>(aux) << "\n";
	}

	return 0;
}
